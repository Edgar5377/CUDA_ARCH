#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iomanip>
#include <assert.h>
#include <iostream>
#include <vector>
#include <chrono>
const int size = 200;
const double imp0 = 377.0;
const int maxTime = 250;

__global__ void updateHy(double* hy, double* ez, int size, double imp0) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size - 1) {
        hy[tid] += (ez[tid + 1] - ez[tid]) / imp0;
    }
}

__global__ void updateEz(double* hy, double* ez, int size, double imp0) {
    int mm = blockIdx.x * blockDim.x + threadIdx.x;
    if (mm >= 1 && mm < size) {
        ez[mm] += (hy[mm] - hy[mm - 1]) * imp0;
    }
}

__global__ void setEzNode(double* ez, int qtime) {
    ez[0] = std::exp(-1.0 * (qtime - 30.0) * (qtime - 30.0) / 100.0);
}


int main()
{    

    std::vector<double> ez(size, 0.0);
    std::vector<double> hy(size, 0.0);
    std::vector<double> E50(maxTime, 0.0);
    std::vector<std::vector<double>> ez_time;
    std::vector<std::vector<double>> hy_time;



   
   
    double* d_ez, * d_hy;
    hipMalloc(&d_ez, size * sizeof(double));
    hipMalloc(&d_hy, size * sizeof(double));

    hipMemcpy(d_ez, ez.data(), size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_hy, hy.data(), size * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocksHy = (size - 1 + blockSize - 1) / blockSize;
    int numBlocksEz = (size + blockSize - 1) / blockSize;


    auto start = std::chrono::system_clock::now(); // Tomar tiempo
    for (int qtime = 1; qtime <= maxTime; ++qtime) {
        // actualiar campo magnetico
        updateHy << <numBlocksHy, blockSize >> > (d_hy, d_ez, size, imp0);
        hipDeviceSynchronize();

        // actualizar campo electrico
        updateEz << <numBlocksEz, blockSize >> > (d_hy, d_ez, size, imp0);
        hipDeviceSynchronize();

        // actualizar ez en el mismo gpu
        setEzNode << <1, 1 >> > (d_ez, qtime);
    }




    auto end = std::chrono::system_clock::now(); //Tomar tiempo 
    std::chrono::duration<float, std::milli> duration = end - start;

    std::cout << "El tiempo total del proceso fue " << duration.count() << "ms" ;

    hipMemcpy(ez.data(), d_ez, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(hy.data(), d_hy, size * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_ez);
    hipFree(d_hy);
   
    /*
    

    
    std::cout << "ez matrix:\n";
    for (const auto& time_step : ez_time) {
        for (const auto& value : time_step) {
            std::cout << std::setprecision(2) << std::setw(10) << value << " ";
        }
        std::cout << "\n";
    }

    std::cout << "hy matrix:\n";
    for (const auto& time_step : hy_time) {
        for (const auto& value : time_step) {
            std::cout << std::setprecision(2) << std::setw(10) << value << " ";
        }
        std::cout << "\n";
    }
    */

    return 0;

}
